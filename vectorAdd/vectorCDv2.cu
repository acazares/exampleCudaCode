
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <hip/hip_complex.h>
#include "hipblas.h"

__global__ void vectorAdd(const hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex cDouble, int numElements)
{
  int blockfst = blockIdx.x * 256;
  int fstglb = blockfst + (threadIdx.x * 2);
  int sndglb = fstglb + 1;

  hipDoubleComplex Afst = A[fstglb];
  hipDoubleComplex Asnd = A[sndglb];
  hipDoubleComplex Bfst = B[fstglb];
  hipDoubleComplex Bsnd = B[sndglb];

  if (fstglb < numElements && sndglb < numElements) {
    B[fstglb] = hipCadd(hipCmul(Afst,cDouble),Bfst);
    B[sndglb] = hipCadd(hipCmul(Asnd,cDouble),Bsnd);
  }
  else if (fstglb < numElements) {
    B[fstglb] = hipCadd(hipCmul(Afst,cDouble),Bfst);
  }
  else if (sndglb < numElements) {
    B[sndglb] = hipCadd(hipCmul(Asnd,cDouble),Bsnd);
  }
}

int main()
{
  int numElements = pow(2,20);
  size_t size = numElements * sizeof(hipDoubleComplex);
    
  hipDoubleComplex *h_A, *h_B; // host pointers
  hipDoubleComplex *d_A, *d_B = NULL; // device pointers
    
  // Allocate space on the host
  hipError_t status_HA = hipHostMalloc((hipDoubleComplex **)&h_A, size, hipHostMallocDefault);
  hipError_t status_HB = hipHostMalloc((hipDoubleComplex **)&h_B, size, hipHostMallocDefault);
    
  // Make sure memory was allocated properly on the host
  if((status_HA || status_HB) != hipSuccess) {
    printf("Memory Allocation Error");
    return 0;
  }
    
  // Allocate space on the device (GPU)
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  
  // Initialize the host input vectors
  for (int b = 0; b < numElements; b++)
    {
      h_A[b] = make_hipDoubleComplex(2.25,2.25);
      h_B[b] = make_hipDoubleComplex(2.25,2.25);
    }
    
  // Copying vectors from host to device
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  
  hipError_t error = hipSuccess;
  
  hipDoubleComplex CDouble = make_hipDoubleComplex(2.25,2.25);
  int blocksize = 128;
  int gridsize = numElements / 256;
  
  vectorAdd <<< gridsize, blocksize >>> (d_A, d_B, CDouble, numElements);
 
  error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "Kernel didn't launch: %d %d %s \n", blocksize, gridsize, hipGetErrorString(error)); 
  }

  // Copy back the results of vector C  
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    
  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  
  // Free host memory
  hipHostFree(h_A);
  hipHostFree(h_B);
  
  hipDeviceReset();
  return 0;
}
