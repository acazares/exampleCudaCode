
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <hip/hip_complex.h>
#include "hipblas.h"
#include <cstdio>

__global__ void vectorAdd(const hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex cDouble, int numElements)
{
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index < numElements){ 
    B[index] = hipCadd(hipCmul(A[index],cDouble),B[index]);
  }
}

int main()
{
  int numElements = pow(2,20);
  size_t size = numElements * sizeof(hipDoubleComplex);
    
  hipDoubleComplex *h_A, *h_B; // host pointers
  hipDoubleComplex *d_A, *d_B = NULL; // device pointers
    
  // Allocate space on the host
  hipError_t status_HA = hipHostMalloc((hipDoubleComplex **)&h_A, size, hipHostMallocDefault);
  hipError_t status_HB = hipHostMalloc((hipDoubleComplex **)&h_B, size, hipHostMallocDefault);
    
  // Make sure memory was allocated properly on the host
  if((status_HA || status_HB) != hipSuccess) {
    printf("Memory Allocation Error");
    return 0;
  }
    
  // Allocate space on the device (GPU)
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  
  // Initialize the host input vectors
  for (int b = 0; b < numElements; b++)
    {
      h_A[b] = make_hipDoubleComplex(2.25,2.25);
      h_B[b] = make_hipDoubleComplex(2.25,2.25);
    }
    
  // Copying vectors from host to device
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  
  hipError_t error = hipSuccess;
  
  int blocksize = 128;
  int blocksPerGrid = ((numElements -1) / blocksize) + 1;
  
  hipDoubleComplex CDouble = make_hipDoubleComplex(2.25,2.25);
  vectorAdd <<< blocksPerGrid, blocksize >>> (d_A, d_B, CDouble, numElements);
  vectorAdd <<< blocksPerGrid, blocksize >>> (d_A, d_B, CDouble, numElements);
  error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "Kernel didn't launch: %d %d %s \n", blocksize, blocksPerGrid, hipGetErrorString(error)); 
  }

  // Copy back the results of vector C  
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    
  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  
  // Free host memory
  hipHostFree(h_A);
  hipHostFree(h_B);
  
  hipDeviceReset();
  return 0;
}
