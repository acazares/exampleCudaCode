
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <hip/hip_complex.h>
#include "hipblas.h"

__global__ void vectorAdd(const hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex cDouble, int numElements)
{
  extern __shared__ hipDoubleComplex SM[];

  int i = threadIdx.x / 2;
  int off = 64 * blockIdx.x;

  if(threadIdx.x % 2 == 0) {
    SM[2*i] = A[off+i];
  }
  else {
    SM[2*i+1] = B[off+i];
  }
  
  __syncthreads();

  if(threadIdx.x % 2 == 0){ 
    B[off+i] = hipCadd(hipCmul(SM[2*i],cDouble),SM[2*i+1]);
  }
}

int main()
{
  int numElements = pow(2,20);
  size_t size = numElements * sizeof(hipDoubleComplex);
    
  hipDoubleComplex *h_A, *h_B; // host pointers
  hipDoubleComplex *d_A, *d_B = NULL; // device pointers
    
  // Allocate space on the host
  hipError_t status_HA = hipHostMalloc((hipDoubleComplex **)&h_A, size, hipHostMallocDefault);
  hipError_t status_HB = hipHostMalloc((hipDoubleComplex **)&h_B, size, hipHostMallocDefault);
    
  // Make sure memory was allocated properly on the host
  if((status_HA || status_HB) != hipSuccess) {
    printf("Memory Allocation Error");
    return 0;
  }
    
  // Allocate space on the device (GPU)
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  
  // Initialize the host input vectors
  for (int b = 0; b < numElements; b++)
    {
      h_A[b] = make_hipDoubleComplex(2.25,2.25);
      h_B[b] = make_hipDoubleComplex(2.25,2.25);
    }
    
  // Copying vectors from host to device
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  
  hipError_t error = hipSuccess;
  
  int blocksize = 128;
  int blocksPerGrid = numElements / 64;

  int SM_size = (blocksize * 2) * (sizeof(hipDoubleComplex));
  
  hipDoubleComplex CDouble = make_hipDoubleComplex(2.25,2.25);

  vectorAdd <<< blocksPerGrid, blocksize, SM_size >>> (d_A, d_B, CDouble, numElements);

  error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "Kernel didn't launch: %d %d %s \n", blocksize, blocksPerGrid, hipGetErrorString(error)); 
  }

  // Copy back the results of vector C  
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    
  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  
  // Free host memory
  hipHostFree(h_A);
  hipHostFree(h_B);
  
  hipDeviceReset();
  return 0;
}
