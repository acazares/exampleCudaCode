#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_complex.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/system_error.h>
#include <math.h>

struct zaxpy_functor {
  const hipDoubleComplex x;

  zaxpy_functor(hipDoubleComplex _x) : x(_x) {}

  __host__ __device__
  hipDoubleComplex operator()(const hipDoubleComplex& A, const hipDoubleComplex& B) const {
    return hipCadd(hipCmul(A,x),B);
  }
};

int main() {

  int numElements = pow(2,20);
  
  thrust::device_vector<hipDoubleComplex> A(numElements);
  thrust::device_vector<hipDoubleComplex> B(numElements);
  hipDoubleComplex CDouble = make_hipDoubleComplex(2.25,2.25);

  thrust::fill(A.begin(), A.end(), CDouble);
  thrust::fill(B.begin(), B.end(), CDouble);
    
  thrust::transform(A.begin(), A.end(), B.begin(), B.begin(), zaxpy_functor(CDouble));

  return 0;
}
