#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <math.h>
#include <thrust/device_vector.h>
#include "hipblas.h"

struct dot_functor {
  
  dot_functor() {}
  
  __host__ __device__
  
  hipDoubleComplex operator()(const hipDoubleComplex& A, const hipDoubleComplex& B) const {
    return hipCmul(hipConj(A),B);
  }
};

struct sum_functor {

  sum_functor() {}

  __host__ __device__

  hipDoubleComplex operator()(const hipDoubleComplex& A, const hipDoubleComplex& B) const {
    return hipCadd(A,B);
  }
};


int main() {
  
  int numElements = pow(2,21);
  
  thrust::device_vector<hipDoubleComplex> A(numElements);
  thrust::device_vector<hipDoubleComplex> B(numElements);
  hipDoubleComplex CDouble = make_hipDoubleComplex(2.25,2.25);
  
  thrust::fill(A.begin(), A.end(), CDouble);
  thrust::fill(B.begin(), B.end(), CDouble);
  
  thrust::transform(A.begin(), A.end(), B.begin(), B.begin(), dot_functor());
  
  hipDoubleComplex result = thrust::reduce(B.begin(), B.end(), (hipDoubleComplex) make_hipDoubleComplex(0.0,0.0), sum_functor());
  
  return 0;
}
