#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <stdio.h>

__global__ void Map(const hipDoubleComplex *A, hipDoubleComplex *B, int numElements) {

  int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index < numElements){
    B[index] = hipCmul(hipConj(A[index]),B[index]);
  }
}

__global__ void reduce(hipDoubleComplex *B_idata, hipDoubleComplex *B_odata) {

  extern __shared__ hipDoubleComplex SM[];

  int tid = threadIdx.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  SM[tid] = B_idata[index];
  __syncthreads();

    
  for(int s = 1; s < blockDim.x; s *= 2) {
    if(tid % (2*s) == 0) {
      SM[tid] = hipCadd(SM[tid],SM[tid + s]);
    }
    __syncthreads();
  }

  if (tid == 0) {
    B_odata[blockIdx.x] = SM[0];
  }
}

__global__ void myZdotc(hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex *O, int numElements) {

  int blocksize = 128;
  int gridsize = ((numElements - 1) / blocksize) + 1;
  int SMsize = blocksize * sizeof(hipDoubleComplex);

  Map <<< gridsize, blocksize >>> (A,B,numElements);
  hipDeviceSynchronize();
  hipDoubleComplex *temp;
  while (gridsize > 0) {
    reduce <<< gridsize, blocksize, SMsize >>>(B,A);
    hipDeviceSynchronize();
    temp = A;
    A = B;
    B = temp;
    gridsize >>= 7;
  }
  O[0] = B[0];
}

int main() 
{
  int numElements = pow(2,21);
  size_t size = numElements * sizeof(hipDoubleComplex);
  size_t size1 = sizeof(hipDoubleComplex);

  hipDoubleComplex *h_A, *h_B, *h_O;
  hipDoubleComplex *d_A, *d_B, *d_O;

  hipHostMalloc((hipDoubleComplex**)&h_A, size);
  hipHostMalloc((hipDoubleComplex**)&h_B, size);
  hipHostMalloc((hipDoubleComplex**)&h_O, size1);

  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_O, size1);

  for(int i = 0; i < numElements; i++)
    {
      h_A[i] = make_hipDoubleComplex(2.25,2.25);
      h_B[i] = make_hipDoubleComplex(2.25,2.25);
    }
  
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_O, h_O, size1, hipMemcpyHostToDevice);
  
  myZdotc <<< 1, 1 >>> (d_A,d_B,d_O,numElements);
  
  hipMemcpy(h_O, d_O, size1, hipMemcpyDeviceToHost);

  std::cout << hipCreal(h_O[0]) << ":" << hipCimag(h_O[0]) <<'\n';

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_O);

  hipHostFree(h_A);
  hipHostFree(h_B);
  hipHostFree(h_O);

  hipDeviceReset();

  return 0;
}
