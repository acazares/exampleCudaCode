
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hip/hip_complex.h>
#include <math.h>
#include <stdio.h>
#include <iostream>


int main() {

  int numElements = pow (2,21);
  size_t size = numElements * sizeof(hipDoubleComplex);

  hipDoubleComplex *h_A, *h_B;
  hipDoubleComplex *d_A, *d_B;

  hipHostMalloc((hipDoubleComplex**) &h_A, size, hipHostMallocDefault);
  hipHostMalloc((hipDoubleComplex**) &h_B, size, hipHostMallocDefault);

  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);

  for (int i = 0; i < numElements; i++) {  
    h_A[i] = make_hipDoubleComplex(2.25,2.25);
    h_B[i] = make_hipDoubleComplex(2.25,2.25);
  }

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipDoubleComplex *result;
  hipHostMalloc((hipDoubleComplex**) &result, sizeof(hipDoubleComplex), hipHostMallocDefault);

  hipblasStatus_t stat = hipblasZdotc(handle, numElements, d_A, 1, d_B, 1, result);

  if (stat == HIPBLAS_STATUS_SUCCESS) {
    std::cout << "The operation completed successfully\n";
  }

  hipFree(d_A);
  hipFree(d_B);

  hipHostFree(h_A);
  hipHostFree(h_B);

  hipDeviceReset();
}
